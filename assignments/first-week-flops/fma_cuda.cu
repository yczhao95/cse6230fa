
extern "C" {
  #include "fma_cuda.h"
}
#include "fma_dev.h"

#define CUDA_CHK(cerr) do {hipError_t _cerr = (cerr); if ((_cerr) != hipSuccess) fprintf(stderr,"[%s, %d] Cuda error %s\n", __FILE__, __LINE__, hipGetErrorString(_cerr)); return 1;} while(0)

int
fma_dev_initialize (int N, int T, int *numDevices, float ***a)
{
  *numDevices = 0;
  *a = NULL;
  return 0;
}

int
fma_dev_free (int N, int T, int *numDevices, float ***a)
{
  *numDevices = 0;
  *a = NULL;
  return 0;
}

int
fma_dev_start (int N, int T, int numDevices, float **a, float b, float c)
{
  return 0;
}

int
fma_dev_end (int N, int T, int numDevices, float **a, float b, float c)
{
  return 0;
}
