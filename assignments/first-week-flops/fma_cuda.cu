#include "hip/hip_runtime.h"

extern "C" {
  #include <stdio.h>
  #include "fma_cuda.h"
}
#include "fma_dev.h"

#define CUDA_CHK(cerr) do {hipError_t _cerr = (cerr); if ((_cerr) != hipSuccess) {fprintf(stderr,"[%s, %d] Cuda error %s\n", __FILE__, __LINE__, hipGetErrorString(_cerr)); return 1;}} while(0)

__global__
static void
fma_initialize (int N, float *a)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int gridSize = gridDim.x * blockDim.x;

  for (int i = tid; i < N; i+= gridSize) {
    a[i] = i;
  }
}

int
fma_dev_initialize (int N, int T, int *numDevices, float ***a)
{
  float **aa = NULL;
  hipError_t cerr;

  cerr = hipGetDeviceCount (numDevices); CUDA_CHK(cerr);
  printf ("numDevices %d\n", *numDevices);
  if (*numDevices) {
    aa = (float **) malloc (*numDevices * sizeof (float *));
    if (!aa) {
      fprintf (stderr, "Failed to allocate aa\n");
      return 1;
    }
    for (int i = 0; i < *numDevices; i++) {
      struct hipDeviceProp_t prop;
      int block, grid;

      cerr = hipSetDevice(i); CUDA_CHK(cerr);
      cerr = hipMalloc (&aa[i], N * sizeof (float)); CUDA_CHK(cerr);
      cerr = hipGetDeviceProperties (&prop, i); CUDA_CHK(cerr);
      block = prop.maxThreadsPerBlock;
      grid = (N + block - 1) / block;
      fma_initialize<<<grid, block>>>(N, aa[i]);
      cerr = hipDeviceSynchronize(); CUDA_CHK(cerr);
    }
  }
  *a = aa;
  return 0;
}

int
fma_dev_free (int N, int T, int *numDevices, float ***a)
{
  hipError_t cerr;

  for (int i = 0; i < *numDevices; i++) {
    cerr = hipSetDevice(i); CUDA_CHK(cerr);
    cerr = hipFree ((*a)[i]); CUDA_CHK(cerr);
  }
  free (*a);
  *a = NULL;
  return 0;
}

int
fma_dev_start (int N, int T, int numDevices, float **a, float b, float c)
{
  hipError_t cerr;

  for (int i = 0; i < numDevices; i++) {
    struct hipDeviceProp_t prop;
    int block, grid;

    cerr = hipSetDevice(i); CUDA_CHK(cerr);
    cerr = hipGetDeviceProperties (&prop, i); CUDA_CHK(cerr);
    block = prop.maxThreadsPerBlock;
    grid = (N + block - 1) / block;
    fma_loop_dev<<<grid, block>>>(N, T, a[i], b, c);
  }
  return 0;
}

int
fma_dev_end (int N, int T, int numDevices, float **a, float b, float c)
{
  hipError_t cerr;

  for (int i = 0; i < numDevices; i++) {
    cerr = hipSetDevice(i); CUDA_CHK(cerr);
    cerr = hipDeviceSynchronize(); CUDA_CHK(cerr);
  }
  return 0;
}
