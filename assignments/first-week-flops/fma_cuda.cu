#include "hip/hip_runtime.h"

extern "C" {
  #include <stdio.h>
  #include "fma_cuda.h"
}
#include "fma_dev.h"

#define CUDA_CHK(cerr) do {hipError_t _cerr = (cerr); if ((_cerr) != hipSuccess) {fprintf(stderr,"[%s, %d] Cuda error %s\n", __FILE__, __LINE__, hipGetErrorString(_cerr)); return 1;}} while(0)

__global__
static void
fma_initialize (int N, float *a)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int gridSize = gridDim.x * blockDim.x;

  for (int i = tid; i < N; i+= gridSize) {
    a[i] = i;
  }
}

int
fma_dev_initialize (int N, int T, int *numDevices, float ***a)
{
  float **aa = NULL;
  int runtimeVersion;
  int driverVersion;
  hipError_t cerr;

  if (!N) {
    *numDevices = 0;
    *a = NULL;
    return 0;
  }
  cerr = hipRuntimeGetVersion (&runtimeVersion);
  if (cerr == hipErrorInsufficientDriver) {
    *numDevices = 0;
    *a = NULL;
    return 0;
  }
  CUDA_CHK(cerr);
  cerr = hipDriverGetVersion (&driverVersion); CUDA_CHK(cerr);
  cerr = hipGetDeviceCount (numDevices); CUDA_CHK(cerr);
  if (*numDevices) {
    aa = (float **) malloc (*numDevices * sizeof (float *));
    if (!aa) {
      fprintf (stderr, "Failed to allocate aa\n");
      return 1;
    }
    for (int i = 0; i < *numDevices; i++) {
      struct hipDeviceProp_t prop;
      int block, grid;

      cerr = hipSetDevice(i); CUDA_CHK(cerr);
      cerr = hipMalloc (&aa[i], N * sizeof (float)); CUDA_CHK(cerr);
      cerr = hipGetDeviceProperties (&prop, i); CUDA_CHK(cerr);
      block = prop.maxThreadsPerBlock;
      grid = (N + block - 1) / block;
      fma_initialize<<<grid, block>>>(N, aa[i]);
      cerr = hipDeviceSynchronize(); CUDA_CHK(cerr);
    }
  }
  *a = aa;
  return 0;
}

int
fma_dev_free (int N, int T, int *numDevices, float ***a)
{
  hipError_t cerr;

  for (int i = 0; i < *numDevices; i++) {
    cerr = hipSetDevice(i); CUDA_CHK(cerr);
    cerr = hipFree ((*a)[i]); CUDA_CHK(cerr);
  }
  free (*a);
  *a = NULL;
  return 0;
}

int
fma_dev_start (int N, int T, int blocksize, int gridsize, int numDevices, float **a, float b, float c)
{
  hipError_t cerr;

  for (int i = 0; i < numDevices; i++) {
    struct hipDeviceProp_t prop;
    int block, grid;

    cerr = hipSetDevice(i); CUDA_CHK(cerr);
    if (blocksize <= 0) {
      cerr = hipGetDeviceProperties (&prop, i); CUDA_CHK(cerr);
      block = prop.maxThreadsPerBlock;
    }
    else {
      block = blocksize;
    }
    if (gridsize <= 0) {
      grid = (N + block - 1) / block;
    }
    else {
      grid = gridsize;
    }
    fma_loop_dev<<<grid, block>>>(N, T, a[i], b, c);
  }
  return 0;
}

int
fma_dev_end (int N, int T, int numDevices, float **a, float b, float c)
{
  hipError_t cerr;

  for (int i = 0; i < numDevices; i++) {
    cerr = hipSetDevice(i); CUDA_CHK(cerr);
    cerr = hipDeviceSynchronize(); CUDA_CHK(cerr);
  }
  return 0;
}
