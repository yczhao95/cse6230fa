#include "hip/hip_runtime.h"

#include "fma_dev.h"

__global__
void
fma_loop_dev (int N, int T, float *a, float b, float c)
{
  int my_thread = threadIdx.x + blockIdx.x * blockDim.x;
  int num_threads = gridDim.x * blockDim.x;

  for (int i = my_thread; i < N; i+= num_threads) {
    for (int j = 0; j < T; j++) {
      a[i] = a[i] * b + c;
    }
  }
}
